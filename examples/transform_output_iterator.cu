#include "hip/hip_runtime.h"
#include <thrust/iterator/iterator_traits.h>
#include <thrust/iterator/iterator_facade.h>
#include <thrust/iterator/iterator_adaptor.h>

#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/set_operations.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>  
#include <iostream>
#include <cstdint>

#define PRINTER(name) print(#name, (name))
template <template <typename...> class V, typename T, typename ...Args>
void print(const char* name, const V<T,Args...> & v)
{
    std::cout << name << ":\t";
    thrust::copy(v.begin(), v.end(), std::ostream_iterator<T>(std::cout, "\t"));
    std::cout << std::endl;
}


template <typename OutputIterator, typename UnaryFunction>
class Proxy
{
    UnaryFunction& fun;
    OutputIterator& out;

public:
    __host__ __device__
    Proxy(UnaryFunction& fun, OutputIterator& out) : fun(fun), out(out) {}

    template <typename T>
    __host__ __device__
    Proxy operator=(const T& x) const
    {
        *out = fun(x);
        return *this;
    }
};


// This iterator is a wrapper around another OutputIterator which
// applies a UnaryFunction before writing to the OutputIterator.
template <typename OutputIterator, typename UnaryFunction>
class transform_output_iterator : public thrust::iterator_adaptor<
                                    transform_output_iterator<OutputIterator, UnaryFunction>
                                                                      , OutputIterator
                                      , thrust::use_default
                                      , thrust::use_default
                                      , thrust::use_default
                                                                      , Proxy<const OutputIterator, const UnaryFunction> >
{
    UnaryFunction fun;

public:

    friend class thrust::iterator_core_access;

    // shorthand for the name of the iterator_adaptor we're deriving from
    typedef thrust::iterator_adaptor<
      transform_output_iterator<OutputIterator, UnaryFunction>,
      OutputIterator, thrust::use_default, thrust::use_default, thrust::use_default, Proxy<const OutputIterator, const UnaryFunction>
    > super_t;

    __host__ __device__
    transform_output_iterator(OutputIterator out, UnaryFunction fun) : super_t(out), fun(fun)
    {
    }


private:
    __host__ __device__
    typename super_t::reference dereference() const
    {
        return Proxy<const OutputIterator, const UnaryFunction>(fun, this->base_reference());
    }
};


struct Multiplier
{
    template<typename Tuple>
    __host__ __device__
    auto operator()(Tuple t) const -> decltype(thrust::get<0>(t) * thrust::get<1>(t))
    {
        return thrust::get<0>(t) * thrust::get<1>(t);
    }
};


template <typename OutputIterator, typename UnaryFunction>
transform_output_iterator<OutputIterator, UnaryFunction>
__host__ __device__
make_transform_output_iterator(OutputIterator out, UnaryFunction fun)
{
    return transform_output_iterator<OutputIterator, UnaryFunction>(out, fun);
}

int main()
{
  int Lkeys[] =   { 1, 2, 4, 5, 6 };
  int Lvals[] =   { 3, 4, 1, 2, 1 };
  int Rkeys[] =   { 1, 3, 4, 5, 6, 7 };
  int Rvals[] =   { 2, 1, 1, 4, 1, 2 };

  size_t Lsize = sizeof(Lkeys)/sizeof(int);
  size_t Rsize = sizeof(Rkeys)/sizeof(int);

  thrust::device_vector<int> Lkeysv(Lkeys, Lkeys+Lsize);
  thrust::device_vector<int> Lvalsv(Lvals, Lvals+Lsize);
  thrust::device_vector<int> Rkeysv(Rkeys, Rkeys+Rsize);
  thrust::device_vector<int> Rvalsv(Rvals, Rvals+Rsize);

  std::size_t min_size = std::min(Lsize, Rsize);

  thrust::device_vector<int> result_keys(min_size);
  thrust::device_vector<int> result_values(min_size);

  auto zipped_values = thrust::make_zip_iterator(thrust::make_tuple(Lvalsv.begin(), Rvalsv.begin()));

  auto output_it = make_transform_output_iterator(result_values.begin(), Multiplier());

  auto result_pair = thrust::set_intersection_by_key(Lkeysv.begin(), Lkeysv.end(), Rkeysv.begin(), Rkeysv.end(), zipped_values, result_keys.begin(), output_it);

  std::size_t new_size = result_pair.first - result_keys.begin();

  result_keys.resize(new_size);
  result_values.resize(new_size);
  PRINTER(result_keys);
  PRINTER(result_values);
}
